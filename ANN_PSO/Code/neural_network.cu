#include "hip/hip_runtime.h"
#include "neural_network.h"
#include <stdio.h>

#define INF 1000000000
//Basic cuda error checking macro
//TODO: Add cuRAND and cuBLAS error checking macros
#define cudaCheckError()\
{\
	hipError_t e = hipGetLastError();\
	if(e != hipSuccess)\
	{\
		printf("CUDA failure: %s%d: %s", __FILE__, __LINE__, hipGetErrorString(e));\
		exit(EXIT_FAILURE);\
	}\
}

using namespace std;

__global__
void Normalize(float *Array, int Number, float MaxValue)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Number)
        Array[Index] = (Array[Index] - 0.5f) * MaxValue;
}

__device__
float ActivationFunction(float Input)
{
    float Activation = 1 / (1 + __expf(-Input));
    return Activation;
}

__global__
void TrainKernel(int Epochs, float *WeightsAndBiases, hiprandState_t *States, int NumParticles, float *FitnessArray)
{
    //Particle index is block index since there is one block for each particle
    int Index = blockIdx.x;

    //Initialize PBest and LBest
    float PersonalBest = INF;
    float LocalBest = INF;
    float Fitness = 0.0f;

    //Set left and right neighbours
	int Left = (NumParticles + Index - 1) % NumParticles;
	int Right = (1 + Index) % NumParticles;

    //Initialize random number generator states
    hiprand_init(Index, Index, 0, &States[Index]);
    hiprandState_t LocalState = States[Index];

    //Initialize c1, c2, chi
    float C1 = 2.05f, C2 = 2.05f;
    float Psi = C1 + C2;
    float Chi = abs(2.0f / (2.0f - Psi - sqrt(Psi * Psi - 4.0f * Psi)));
    // printf("PSI: %.5f\t\tCHI: %.5f\n", Psi, Chi);
	float R1, R2;

    //For each epoch
    for(int i = 0; i < Epochs; i++)
    {
        Fitness = 0.0f;

        //Calculate fitness, i.e. loss (MSE?)
        //Main feed forward work to be done here


        if(Fitness < PersonalBest)
        {
            PersonalBest = Fitness;
            //Copy personal best weights and biases
            //Another small kernel launch/iteration?
        }

        //Update local best particle index (left or right)
        // if(PersonalBest > PersonalBestArray[Left])
        //     LocalBest = Left;
        // if(PersonalBest > PersonalBestArray[Right])
        //     LocalBest = Right;

        //Update weights and biases of each particle
    }

    //Find the global best particle (here? or defer to CPU?)
}

NeuralNetwork::NeuralNetwork(int InputNeurons, int HiddenLayers, int HiddenNeurons, int OutputNeurons, int NumParticles)
{
    //NN hyperparameters
    this->InputNeurons = InputNeurons;
    this->HiddenLayers = HiddenLayers;
    this->HiddenNeurons = HiddenNeurons;
    this->OutputNeurons = OutputNeurons;
    this->NumParticles = NumParticles;
    cout << "HYPERPARAMETERS SET" << endl;

    //Initialize random weights and biases on the GPU
    //Calculate total number of weights and biases for memory allocation
    int TotalWeightsAndBiasesPerParticle = ((InputNeurons + 1) * HiddenNeurons)
                                    + (((HiddenNeurons +1) * HiddenNeurons)
                                        * (HiddenLayers - 1))
                                    + ((HiddenNeurons + 1) * OutputNeurons);

    //Total
    int TotalWeightsAndBiases = NumParticles * TotalWeightsAndBiasesPerParticle;

    cout << "TOTAL SPACE FOR WEIGHTS AND BIASES: " << TotalWeightsAndBiases * 4 / 1024 << "KB" << endl;

    //Allocate device memory for weights and biases
    float *WeightsAndBiases;
    hipMalloc((void**)&WeightsAndBiases, TotalWeightsAndBiases * sizeof(float));
    cout << "GPU SPACE ALLOCATED FOR WEIGHTS AND BIASES" << endl;

    //Allocate device memory for velocities
    float *Velocities;
    hipMalloc((void**)&Velocities, TotalWeightsAndBiases * sizeof(float));
    cout << "GPU SPACE ALLOCATED FOR VELOCITIES" << endl;

    //Allocate device memory for velocities
    float *FitnessArray;
    hipMalloc((void**)&FitnessArray, NumParticles * sizeof(float));
    this->FitnessArray = FitnessArray;
    cout << "GPU SPACE ALLOCATED FOR FITNESS VALUES" << endl;

    //Initialize generator
    hiprandGenerator_t Gen;
	hiprandCreateGenerator(&Gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(Gen, time(NULL));
    cout << "CURAND GENERATOR INITIALIZED" << endl;

    //Dim3 variables for Normalize kernel
    dim3 Grid(TotalWeightsAndBiasesPerParticle, 1, 1);
    dim3 Block(NumParticles, 1, 1);

    //Generate weights and biases
    hiprandGenerateUniform(Gen, WeightsAndBiases, TotalWeightsAndBiases);
    Normalize <<<Grid, Block>>> (WeightsAndBiases, TotalWeightsAndBiases, 3.0f);
    this->WeightsAndBiases = WeightsAndBiases;
    cout << "WEIGHTS AND BIASES INITIALIZED ON GPU" << endl;

    //Generate velocities
    hiprandGenerateUniform(Gen, Velocities, TotalWeightsAndBiases);
    Normalize <<<Grid, Block>>> (Velocities, TotalWeightsAndBiases, 5.0f);
    this->Velocities = Velocities;
    cout << "VELOCITIES INITIALIZED ON GPU" << endl;

    //Allocate space for hiprand states
    hiprandState_t *States;
    hipMalloc((void**)&States, NumParticles * sizeof(hiprandState_t));
    this->States = States;
    cout << "SPACE ALLOCATED FOR CURAND STATES" << endl;

    // float* temp = new float[TotalWeightsAndBiases];
    // hipMemcpy(temp, WeightsAndBiases, TotalWeightsAndBiases * sizeof(float), hipMemcpyDeviceToHost);
    // for(int i = 0; i < TotalWeightsAndBiases; i++)
    //     cout << temp[i] << endl;

    //Weights and Biases to be stored in a single contiguous array
    //In Column Major format(?) (Verify) (Not necessary since only random init taking place here)

    //Pointers to required positions
    this->InputHidden = this->WeightsAndBiases;
    this->HiddenHidden = this->InputHidden
                        + ((InputNeurons + 1) * HiddenNeurons) * NumParticles;
    this->HiddenOutput = this->HiddenHidden
                        + (((HiddenNeurons +1) * HiddenNeurons)
                        * (HiddenLayers - 1)) * NumParticles;
    cout << "LAYER POINTERS SET" << endl;

    //Synchronize all kernel calls upto this point
    hipDeviceSynchronize();
}

// NeuralNetwork::Load()
// Loads the input feature vectors into an array on the CPU and transfers it to
// the GPU. Method of transferring and thus training (with or without streams)
// will vary depending upon the size of input data.
void NeuralNetwork::Load(const char *FileName)
{
    int Size;
    float *InputFeatures;
    float *OutputFeatures;
    int Width = this->InputNeurons;
    fstream FIn;
    FIn.open(FileName);
    if(!FIn.fail())
    {
        cout << "FILE OPENED" << endl;
        FIn >> Size;
        InputFeatures = new float[Size * Width];
        OutputFeatures = new float[Size];
        cout << "SPACE ALLOCATED" << endl;
        int temp;

        for(int i = 0; i < Size; i++)
        {
            for(int j = 0; j < Width; j++)
            {
                FIn >> temp;
                InputFeatures[i * Width + j] = float(temp);
            }
            FIn >> temp;
            OutputFeatures[i] = float(temp);
        }
    }
    FIn.close();

    cout << "INPUT OUTPUT SPACE REQUIRED: " << Size * 24 / 1024 << "KB" << endl;

    cout << "INPUT AND OUTPUT LOADED AND FILE CLOSED" << endl;

    //Transfer to GPU (Single hipMemcpy() for the time being)
    float* DeviceInputFeatures;
    hipMalloc((void**)&DeviceInputFeatures, Size * Width * sizeof(float));
    hipMemcpy(DeviceInputFeatures, InputFeatures, Size * Width * sizeof(float), hipMemcpyHostToDevice);
    this->InputFeatures = DeviceInputFeatures;

    float* DeviceOutputFeatures;
    hipMalloc((void**)&DeviceOutputFeatures, Size * sizeof(float));
    hipMemcpy(DeviceOutputFeatures, OutputFeatures, Size * sizeof(float), hipMemcpyHostToDevice);
    this->OutputFeatures = DeviceOutputFeatures;

    cout << "INPUT AND OUTPUT TRANSFERRED TO GPU" << endl;
}

// NeuralNetwork::Train()
// Trains the network using PSO and a set number of particles in order to eliminate
// backpropogation.
void NeuralNetwork::Train(int Epochs)
{
    dim3 Grid(this->NumParticles, 1, 1);
    dim3 Block(1, 1, 1);

    cout << "GRID AND BLOCK SIZE INITIALIZED" << endl;

    //Allocate space for each particle's fitness
    float *Fitness;
    hipMalloc((void**)&Fitness, this->NumParticles);

    //Training kernel
    TrainKernel <<<Grid, Block>>> (Epochs, this->WeightsAndBiases, this->States, this->NumParticles, this->FitnessArray);
    hipDeviceSynchronize();
}
