#include "hip/hip_runtime.h"
#include "neural_network.h"
#define TILE_WIDTH 16
#define INF 1000000000.0f
//Basic cuda error checking macro
//TODO: Add cuRAND and cuBLAS error checking macros
//TODO: Wrap all calls in relevant error checking macros
#define cudaCheckError()\
{\
	hipError_t e = hipGetLastError();\
	if(e != hipSuccess)\
	{\
		printf("CUDA failure: %s%d: %s", __FILE__, __LINE__, hipGetErrorString(e));\
		exit(EXIT_FAILURE);\
	}\
}

using namespace std;

typedef struct PSOParameters
{
	hiprandState_t *States = NULL;
	int NumParticles = 0;
	float *FitnessArray = NULL;
	float *PersonalBestWeights = NULL;
	float *Velocities = NULL;
	float C1 = 0.0f;
	float C2 = 0.0f;
	float XMax = 0.0f;
	float VMax = 0.0f;
} PSOParameters;

typedef struct NNParameters
{
	int Epochs = 0;
	int InputNeurons = 0;
	int HiddenLayers = 0;
	int HiddenNeurons = 0;
	int OutputNeurons = 0;
	int NetworkSize = 0;
	int MaxIOLength = 0;
	int NumVectors = 0;
	float *WeightsAndBiases = NULL;
	float *InputFeatures = NULL;
	float *IntermediateIO = NULL;
	float *OutputFeatures = NULL;
} NNParameters;

// Normalizes a vector to [-MaxValue, MaxValue]
__global__
void Normalize(float *Array, int Number, float MaxValue)
{
    int Index = blockDim.x * blockIdx.x + threadIdx.x;
    if(Index < Number)
        Array[Index] = 2 * (Array[Index] - 0.5f) * MaxValue;
}

// Transpose a matrix
__global__
void Transpose(float *InputMatrix, float *OutputMatrix, int Rows, int Columns)
{
	int IdX = blockDim.x * blockIdx.x + threadIdx.x;
	int IdY = blockDim.y * blockIdx.y + threadIdx.y;
	int TX = threadIdx.x;
	int TY = threadIdx.y;

	__shared__ float Tile[TILE_WIDTH][TILE_WIDTH];

	if(IdX < Columns && IdY < Rows)
	{
		Tile[TX][TY] = InputMatrix[IdX + Columns * IdY];
		OutputMatrix[IdY + Rows * IdX] = Tile[TX][TY];
	}
}

// Small kernel for device to device memory transfers
__global__
void DeviceToDevice(float *Destination, float *Source, int Size)
{
	int Index = blockIdx.x * blockDim.x + threadIdx.x;
	if(Index < Size)
		Destination[Index] = Source[Index];
}

// ReLU activation function
__global__
void ReLU(float *Input, int Size)
{
	int Index = blockIdx.x * blockDim.x + threadIdx.x;
	if(Index < Size)
	{
		if(Input[Index] < 0.0f)
			Input[Index] = 0.001 * Input[Index];
	}
    	// Input[Index] = (1 / (1 + __expf(-Input[Index])));
}

__global__
void Sigmoid(float *Input, int Size)
{
	int Index = blockIdx.x * blockDim.x + threadIdx.x;
	if(Index < Size)
    	Input[Index] = (1 / (1 + __expf(-Input[Index])));
}

// Kernel which actually trains the data.
__global__
void TrainKernel(NNParameters NNParams, PSOParameters PSOParams)
{
    //Particle index is block index since there is one block for each particle
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
	int Id = 0;

	if(Index < PSOParams.NumParticles)
	{
		//Initialize PBest, LBest and fitness
	    float PersonalBest = INF;
		float PersonalBestX = INF;
		int LocalBestIndex = Index;
		float LocalBestX = INF;
	    float Fitness = 0.0f;

	    //Initialize chi, declare r1, r2
	    float Psi = PSOParams.C1 + PSOParams.C2;
	    float Chi = abs(2.0f / (2.0f - Psi - sqrt(Psi * Psi - 4.0f * Psi)));
		float R1, R2;

	    //Set left and right neighbours
		int Left = (PSOParams.NumParticles + Index - 1) % PSOParams.NumParticles;
		int Right = (1 + Index) % PSOParams.NumParticles;

	    //Initialize random number generator states
	    hiprand_init(Index, Index, 0, &PSOParams.States[Index]);
	    hiprandState_t LocalState = PSOParams.States[Index];

		//Pointer to weights and biases
		float *WeightsAndBiases = &NNParams.WeightsAndBiases[Index * NNParams.NetworkSize];
		float *PersonalBestWeights = &PSOParams.PersonalBestWeights[Index * NNParams.NetworkSize];

		//Input, output, matrix and temporary pointers
		float *Input;
		float *Output;
		float *Matrix;
		float *Temp;

		//Grid and block for network sized transfers
		dim3 NetworkGrid(NNParams.NetworkSize / 256 + 1, 1, 1);
		dim3 NetworkBlock(256, 1, 1);

		//cuBLAS handle initialization
		hipblasHandle_t Handle;
		hipblasCreate(&Handle);

		//Alpha and beta values
		float Alpha = 1.0f;
	    float Beta = 0.0f;

	    //For each epoch
	    for(int k = 0; k < NNParams.Epochs; k++)
	    {
			Fitness = 0.0f;

			//Main feed forward work to be done here
			//Calculate fitness, i.e. loss (MSE?)
	        for(int j = 0; j < NNParams.NumVectors; j++)
			{
				//Input hidden multiplication + biases
				Input = &NNParams.InputFeatures[NNParams.InputNeurons * j];
				Output = &NNParams.IntermediateIO[NNParams.MaxIOLength * Index];
				Matrix = &NNParams.WeightsAndBiases[NNParams.NetworkSize * Index];

				hipblasSgemv(Handle, HIPBLAS_OP_N,
					NNParams.HiddenNeurons, NNParams.InputNeurons, &Alpha,
					Matrix, NNParams.HiddenNeurons, Input, 1, &Beta, Output, 1);
				hipDeviceSynchronize();

				Matrix += NNParams.InputNeurons * NNParams.HiddenNeurons;

				//Add biases
				hipblasSaxpy(Handle, NNParams.HiddenNeurons,
					&Alpha, Matrix, 1, Output, 1);
				hipDeviceSynchronize();

				//Activation function
				ReLU <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.HiddenNeurons);
				hipDeviceSynchronize();

				Input = Output + NNParams.MaxIOLength / 2;
				Matrix += NNParams.HiddenNeurons;

				//Hidden hidden loop
				for(int c = 1; c < NNParams.HiddenLayers; c++)
				{
					//Swap input and output
					Temp = Input;
					Input = Output;
					Output = Temp;

					//Multiply
					hipblasSgemv(Handle, HIPBLAS_OP_N,
						NNParams.HiddenNeurons, NNParams.HiddenNeurons, &Alpha,
						Matrix, NNParams.HiddenNeurons, Input, 1, &Beta, Output, 1);
					hipDeviceSynchronize();

					Matrix += NNParams.HiddenNeurons * NNParams.HiddenNeurons;

					//Add biases
					hipblasSaxpy(Handle, NNParams.HiddenNeurons,
						&Alpha, Matrix, 1, Output, 1);
					hipDeviceSynchronize();

					//Activation function
					ReLU <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.HiddenNeurons);
					hipDeviceSynchronize();

					Matrix += NNParams.HiddenNeurons;
				}

				//Hidden output multiplication + biases
				//Multiply
				hipblasSgemv(Handle, HIPBLAS_OP_N,
					NNParams.OutputNeurons, NNParams.HiddenNeurons, &Alpha,
					Matrix, NNParams.OutputNeurons, Input, 1, &Beta, Output, 1);
				hipDeviceSynchronize();

				Matrix += NNParams.HiddenNeurons * NNParams.OutputNeurons;

				//Add biases
				hipblasSaxpy(Handle, NNParams.OutputNeurons,
					&Alpha, Matrix, 1, Output, 1);
				hipDeviceSynchronize();

				//Activation function
				Sigmoid <<<(NNParams.HiddenNeurons - 1) / 32 + 1, 32>>> (Output, NNParams.OutputNeurons);
				hipDeviceSynchronize();

				Fitness += (NNParams.OutputFeatures[j] - Output[0]) * (NNParams.OutputFeatures[j] - Output[0]);
			}

			Fitness /= NNParams.NumVectors;
			__syncthreads();

			//Compare fitness to personal best so far
	        if(Fitness < PersonalBest)
	        {
				//Copy personal best values
	            PersonalBest = Fitness;
				PSOParams.FitnessArray[Index] = Fitness;
	            //Copy personal best weights and biases
				//Device to device transfer
				DeviceToDevice <<<NetworkGrid, NetworkBlock>>> (PersonalBestWeights, WeightsAndBiases, NNParams.NetworkSize);
				hipDeviceSynchronize();
	        }
			__syncthreads();
	        //Update local best particle index (left or right)
	        if(PersonalBest > PSOParams.FitnessArray[Left])
	            LocalBestIndex = Left;
	        if(PersonalBest > PSOParams.FitnessArray[Right])
	            LocalBestIndex = Right;
			__syncthreads();

	        //Update weights and biases of each particle
			for (int i = 0; i < NNParams.NetworkSize; i++)
			{
				//Set index at which position needs to be updated
				Id = Index * NNParams.NetworkSize + i;

				//Set local best and personal best X (weights / biases)
				LocalBestX = PSOParams.PersonalBestWeights[LocalBestIndex * NNParams.NetworkSize + i];
				PersonalBestX = PSOParams.PersonalBestWeights[Index * NNParams.NetworkSize + i];

				//Generate random numbers
				R1 = hiprand_uniform(&LocalState);
				R2 = hiprand_uniform(&LocalState);

				//Update the velocity
				PSOParams.Velocities[Id] = Chi * (PSOParams.Velocities[Id] +
										PSOParams.C1 * R1 * (PersonalBestX - NNParams.WeightsAndBiases[Id]) +
										PSOParams.C2 * R2 * (LocalBestX - NNParams.WeightsAndBiases[Id]));

				//Ensure velocity values are within range
				if (PSOParams.Velocities[Id] > PSOParams.VMax)
					PSOParams.Velocities[Id] = PSOParams.VMax;
				if (PSOParams.Velocities[Id] < -PSOParams.VMax)
					PSOParams.Velocities[Id] = -PSOParams.VMax;

				//Update the position
				NNParams.WeightsAndBiases[Id] = NNParams.WeightsAndBiases[Id] + PSOParams.Velocities[Id];

				// Ensure position values are within range
				if (NNParams.WeightsAndBiases[Id] > PSOParams.XMax)
				{
					NNParams.WeightsAndBiases[Id] = PSOParams.XMax;
					PSOParams.Velocities[Id] = 0.0f;
				}
				if (NNParams.WeightsAndBiases[Id] < -PSOParams.XMax)
				{
					NNParams.WeightsAndBiases[Id] = -PSOParams.XMax;
					PSOParams.Velocities[Id] = 0.0f;
				}
				__syncthreads();
			}
	    }
	}
}

void NeuralNetwork::CheckKernel()
{
	float *a = new float[12];
	float *b = new float[12];

	for(int i = 0; i < 3; i++)
	{
		for(int j = 0; j < 4; j++)
		{
			a[i * 4 + j] = i * 4 + j;
			cout << a[i * 4 + j] << " ";
		}
		cout << endl;
	}

	float *deva, *devb;
	hipMalloc((void**)&deva, 12 * sizeof(float));
	hipMalloc((void**)&devb, 12 * sizeof(float));

	hipMemcpy(deva, a, 12 * sizeof(float), hipMemcpyHostToDevice);
	dim3 Grid((4 - 1) / TILE_WIDTH + 1, (3 - 1) / TILE_WIDTH + 1, 1);
	dim3 Block(TILE_WIDTH, TILE_WIDTH, 1);
	Transpose <<<Grid, Block>>> (deva, devb, 3, 4);

	hipMemcpy(b, devb, 12 * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < 4; i++)
	{
		for(int j = 0; j < 3; j++)
		{
			cout << b[i * 3 + j] << " ";
		}
		cout << endl;
	}
}

//NeuralNetwork::NeuralNetwork()
// Constructor of the NeuralNetwork class
NeuralNetwork::NeuralNetwork(int InputNeurons, int HiddenLayers, int HiddenNeurons, int OutputNeurons, int NumParticles)
{
    //NN hyperparameters
    this->InputNeurons = InputNeurons;
    this->HiddenLayers = HiddenLayers;
    this->HiddenNeurons = HiddenNeurons;
    this->OutputNeurons = OutputNeurons;
    this->NumParticles = NumParticles;
    cout << "HYPERPARAMETERS SET" << endl;

    //Initialize random weights and biases on the GPU
    //Calculate total number of weights and biases for memory allocation
    int NetworkSize = ((InputNeurons + 1) * HiddenNeurons)
                                    + (((HiddenNeurons +1) * HiddenNeurons)
                                        * (HiddenLayers - 1))
                                    + ((HiddenNeurons + 1) * OutputNeurons);
	this->NetworkSize = NetworkSize;

    //Total
    int TotalWeightsAndBiases = NumParticles * NetworkSize;

    cout << "TOTAL SPACE FOR WEIGHTS AND BIASES: " << TotalWeightsAndBiases * 4 / 1024 << "KB" << endl;

    //Allocate device memory for weights and biases
    float *WeightsAndBiases;
    hipMalloc((void**)&WeightsAndBiases, TotalWeightsAndBiases * sizeof(float));
    cout << "GPU SPACE ALLOCATED FOR WEIGHTS AND BIASES" << endl;

	//Allocate device memory for weights and biases
    float *PersonalBestWeights;
    hipMalloc((void**)&PersonalBestWeights, TotalWeightsAndBiases * sizeof(float));
    cout << "GPU SPACE ALLOCATED FOR PERSONAL BEST WEIGHTS AND BIASES" << endl;

	//Max space to be allocated to intermediate I/O
	int MaxIOLength = 2 * max(InputNeurons, max(HiddenNeurons, OutputNeurons));
	this->MaxIOLength = MaxIOLength;
	float *IntermediateIO;
	hipMalloc((void**)&IntermediateIO, MaxIOLength * sizeof(float) * this->NumParticles);
	this->IntermediateIO = IntermediateIO;

    //Allocate device memory for velocities
    float *Velocities;
    hipMalloc((void**)&Velocities, TotalWeightsAndBiases * sizeof(float));
    cout << "GPU SPACE ALLOCATED FOR VELOCITIES" << endl;

    //Allocate device memory for velocities
    float *FitnessArray;
    hipMalloc((void**)&FitnessArray, NumParticles * sizeof(float));
    this->FitnessArray = FitnessArray;
    cout << "GPU SPACE ALLOCATED FOR FITNESS VALUES" << endl;

    //Initialize generator
    hiprandGenerator_t Gen;
	hiprandCreateGenerator(&Gen, HIPRAND_RNG_QUASI_SOBOL32);
	hiprandSetQuasiRandomGeneratorDimensions(Gen, this->NetworkSize);
	hiprandSetPseudoRandomGeneratorSeed(Gen, time(NULL));
	// hiprandCreateGenerator(&Gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    cout << "CURAND GENERATOR INITIALIZED" << endl;

    //Dim3 variables for Normalize kernel
    dim3 Grid(NetworkSize, 1, 1);
    dim3 Block(NumParticles, 1, 1);

    //Generate weights and biases
    hiprandGenerateUniform(Gen, WeightsAndBiases, TotalWeightsAndBiases);
    Normalize <<<Grid, Block>>> (WeightsAndBiases, TotalWeightsAndBiases, 10.0f);
	dim3 TransposeGrid((this->NumParticles - 1) / TILE_WIDTH + 1, (this->NetworkSize - 1) / TILE_WIDTH + 1, 1);
	dim3 TransposeBlock(TILE_WIDTH, TILE_WIDTH, 1);
	Transpose <<<TransposeGrid, TransposeBlock>>> (WeightsAndBiases, PersonalBestWeights, this->NetworkSize, this->NumParticles);
    this->WeightsAndBiases = WeightsAndBiases;
    cout << "WEIGHTS AND BIASES INITIALIZED ON GPU" << endl;

	//Copy generated weights and biases to personal best array for initialization
	DeviceToDevice <<<Grid, Block>>> (WeightsAndBiases, PersonalBestWeights, TotalWeightsAndBiases);
	this->PersonalBestWeights = PersonalBestWeights;

    //Generate velocities
    hiprandGenerateUniform(Gen, Velocities, TotalWeightsAndBiases);
    Normalize <<<Grid, Block>>> (Velocities, TotalWeightsAndBiases, 1.0f);
    this->Velocities = Velocities;
    cout << "VELOCITIES INITIALIZED ON GPU" << endl;

    //Allocate space for hiprand states
    hiprandState_t *States;
    hipMalloc((void**)&States, NumParticles * sizeof(hiprandState_t));
    this->States = States;
    cout << "SPACE ALLOCATED FOR CURAND STATES" << endl;

    // float* temp = new float[TotalWeightsAndBiases];
    // hipMemcpy(temp, WeightsAndBiases, TotalWeightsAndBiases * sizeof(float), hipMemcpyDeviceToHost);
    // for(int i = 0; i < TotalWeightsAndBiases; i++)
    //     cout << temp[i] << endl;

    //Synchronize all kernel calls upto this point
    hipDeviceSynchronize();
}

// NeuralNetwork::Load()
// Loads the input feature vectors into an array on the CPU and transfers it to
// the GPU. Method of transferring and thus training (with or without streams)
// will vary depending upon the size of input data.
void NeuralNetwork::Load(const char *FileName)
{
    int Size;
    float *InputFeatures;
    float *OutputFeatures;
    int Width = this->InputNeurons;
    fstream FIn;
    FIn.open(FileName);
    if(!FIn.fail())
    {
        cout << "FILE OPENED" << endl;
        FIn >> Size;
        InputFeatures = new float[Size * Width];
        OutputFeatures = new float[Size];
        cout << "SPACE ALLOCATED" << endl;
        int temp;

        for(int i = 0; i < Size; i++)
        {
            for(int j = 0; j < Width; j++)
            {
                FIn >> temp;
                InputFeatures[i * Width + j] = float(temp);
            }
            FIn >> temp;
            OutputFeatures[i] = float(temp);
        }
    }
    FIn.close();

    cout << "INPUT OUTPUT SPACE REQUIRED: " << Size * 24 / 1024 << "KB" << endl;
	this->NumVectors = Size;

    cout << "INPUT AND OUTPUT LOADED AND FILE CLOSED" << endl;

    //Transfer to GPU (Single hipMemcpy() for the time being)
    float* DeviceInputFeatures;
    hipMalloc((void**)&DeviceInputFeatures, Size * Width * sizeof(float));
    hipMemcpy(DeviceInputFeatures, InputFeatures, Size * Width * sizeof(float), hipMemcpyHostToDevice);
    this->InputFeatures = DeviceInputFeatures;

    float* DeviceOutputFeatures;
    hipMalloc((void**)&DeviceOutputFeatures, Size * sizeof(float));
    hipMemcpy(DeviceOutputFeatures, OutputFeatures, Size * sizeof(float), hipMemcpyHostToDevice);
    this->OutputFeatures = DeviceOutputFeatures;

    cout << "INPUT AND OUTPUT TRANSFERRED TO GPU" << endl;
}

// NeuralNetwork::Train()
// Trains the network using PSO and a set number of particles in order to eliminate
// backpropogation.
// Assumes weight matrix to be in column major format.
void NeuralNetwork::Train(int Epochs, const char *WeightsFile)
{
	//One block per particle and one thread per block TODO: change
    dim3 Grid((this->NumParticles - 1) / 32 + 1, 1, 1);
    dim3 Block(32, 1, 1);

	//NN parameters struct
	NNParameters NNParams;
	NNParams.Epochs = Epochs;
	NNParams.InputNeurons = this->InputNeurons;
	NNParams.HiddenLayers = this->HiddenLayers;
	NNParams.HiddenNeurons = this->HiddenNeurons;
	NNParams.OutputNeurons = this->OutputNeurons;
	NNParams.NetworkSize = this->NetworkSize;
	NNParams.MaxIOLength = this->MaxIOLength;
	NNParams.NumVectors = this->NumVectors;
	NNParams.InputFeatures = this->InputFeatures;
	NNParams.IntermediateIO = this->IntermediateIO;
	NNParams.OutputFeatures = this->OutputFeatures;
	NNParams.WeightsAndBiases = this->WeightsAndBiases;

	//PSO parameters struct
	PSOParameters PSOParams;
	PSOParams.NumParticles = this->NumParticles;
	PSOParams.C1 = 2.05f;
	PSOParams.C2 = 2.05f;
	PSOParams.XMax = 10.0f;
	PSOParams.VMax = 1.0f;
	PSOParams.FitnessArray = this->FitnessArray;
	PSOParams.States = this->States;
	PSOParams.PersonalBestWeights = this->PersonalBestWeights;
	PSOParams.Velocities = this->Velocities;

    //Training kernel
    TrainKernel <<<Grid, Block>>> (NNParams, PSOParams);
    hipDeviceSynchronize();

	int *InputValues;
	int *HostIndices = new int[this->NumParticles];
	for(int i = 0; i < this->NumParticles; i++)
		HostIndices[i] = i;
	hipMalloc((void**)&InputValues, this->NumParticles * sizeof(int));
	hipMemcpy(InputValues, HostIndices, this->NumParticles * sizeof(int), hipMemcpyHostToDevice);

	int *OutputValues;
	hipMalloc((void**)&OutputValues, this->NumParticles * sizeof(int));

	float *TempCopies = new float[this->NumParticles];
	hipMemcpy(TempCopies, this->FitnessArray, this->NumParticles * sizeof(float), hipMemcpyDeviceToHost);

	// for(int i = 0; i < this->NumParticles; i++)
	// 	cout << i << "\t" << TempCopies[i] << endl;
	// cout << endl;

	float *HostFitness = new float(INF);
	int *HostIndex = new int(INF);

	//Thrust reduce by key
	thrust::stable_sort_by_key(thrust::device, this->FitnessArray, this->FitnessArray + this->NumParticles, InputValues);
	hipDeviceSynchronize();
	hipMemcpy(HostIndex, InputValues, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(HostFitness, this->FitnessArray, sizeof(int), hipMemcpyDeviceToHost);

	cout << "BEST PARTICLE: " << *HostIndex << endl;
	cout << "BEST FITNESS: " << *HostFitness << endl;

	float *DeviceBestNetwork = &this->PersonalBestWeights[this->NetworkSize * (*HostIndex)];
	float *BestNetwork = new float[this->NetworkSize];
	hipMemcpy(BestNetwork, DeviceBestNetwork, this->NetworkSize * sizeof(float), hipMemcpyDeviceToHost);

	//Dump to file
	fstream FOut;
    FOut.open(WeightsFile, fstream::out);
    if(!FOut.fail())
	{
		FOut << this->InputNeurons << endl;
		FOut << this->HiddenLayers << endl;
		FOut << this->HiddenNeurons << endl;
		FOut << this->OutputNeurons << endl;
		for(int i = 0; i < this->NetworkSize; i++)
		{
			FOut << BestNetwork[i] << endl;
		}
	}
	FOut.close();
}

// NeuralNetwork::Test()
// Tests a set of weights and biases and reports the loss
void NeuralNetwork::Test(const char *TestFile, const char *WeightsFile)
{
	fstream FIn;
	int InputNeurons = 0;
	int HiddenLayers = 0;
	int HiddenNeurons = 0;
	int OutputNeurons = 0;
	int NetworkSize = 0;
	float *Weights;
	FIn.open(WeightsFile, fstream::in);
	if(!FIn.fail())
	{
		FIn >> InputNeurons;
		FIn >> HiddenLayers;
		FIn >> HiddenNeurons;
		FIn >> OutputNeurons;

		NetworkSize = ((InputNeurons + 1) * HiddenNeurons)
                            + (((HiddenNeurons +1) * HiddenNeurons)
                                * (HiddenLayers - 1))
                            + ((HiddenNeurons + 1) * OutputNeurons);

		Weights = new float[NetworkSize];
		for(int i = 0; i < NetworkSize; i++)
			FIn >> Weights[i];
	}
	FIn.close();

	int NumSamples = 0;
	float *InputFeatures;
	float *OutputFeatures;
	FIn.open(TestFile, fstream::in);
	if(!FIn.fail())
	{
		FIn >> NumSamples;
		InputFeatures = new float[NumSamples * InputNeurons];
		OutputFeatures = new float[NumSamples];

		for(int i = 0; i < NumSamples; i++)
		{
			for(int j = 0; j < InputNeurons; j++)
				FIn >> InputFeatures[i * InputNeurons + j];

			FIn >> OutputFeatures[i];
		}
	}
	FIn.close();

	float *InputVectors;
	hipMalloc((void**)&InputVectors, NumSamples * InputNeurons * sizeof(float));
	hipMemcpy(InputVectors, InputFeatures, NumSamples * InputNeurons * sizeof(float), hipMemcpyHostToDevice);

	// float *OutputVector;
	// hipMalloc((void**)&OutputVector, NumSamples * sizeof(float));
	// hipMemcpy(OutputVector, OutputFeatures, NumSamples * sizeof(float), hipMemcpyHostToDevice);

	float *WeightsAndBiases;
	hipMalloc((void**)&WeightsAndBiases, NetworkSize * sizeof(float));
	hipMemcpy(WeightsAndBiases, Weights, NetworkSize * sizeof(float), hipMemcpyHostToDevice);

	hipblasHandle_t Handle;
	hipblasCreate(&Handle);

	float Alpha = 1.0f, Beta = 0.0f;
	float Fitness = 0.0f, TempFitness = 0.0f;
	float *Input, *Output, *Matrix, * Temp;

	int MaxIOLength = 2 * max(InputNeurons, max(HiddenNeurons, OutputNeurons));
	float *IntermediateIO;
	hipMalloc((void**)&IntermediateIO, MaxIOLength * sizeof(float));

	//Main feed forward work to be done here
	//Calculate fitness, i.e. loss (MSE?)
	for(int j = 0; j < NumSamples; j++)
	{
		//Input hidden multiplication + biases
		Input = &InputVectors[InputNeurons * j];
		Output = IntermediateIO;
		Matrix = WeightsAndBiases;

		hipblasSgemv(Handle, HIPBLAS_OP_N,
			HiddenNeurons, InputNeurons, &Alpha,
			Matrix, HiddenNeurons, Input, 1, &Beta, Output, 1);
		hipDeviceSynchronize();

		Matrix += InputNeurons * HiddenNeurons;

		//Add biases
		hipblasSaxpy(Handle, HiddenNeurons,
			&Alpha, Matrix, 1, Output, 1);
		hipDeviceSynchronize();

		//Activation function
		ReLU <<<(HiddenNeurons - 1) / 32 + 1, 32>>> (Output, HiddenNeurons);
		hipDeviceSynchronize();

		Input = Output + MaxIOLength / 2;
		Matrix += HiddenNeurons;

		//Hidden hidden loop
		for(int c = 1; c < HiddenLayers; c++)
		{
			//Swap input and output
			Temp = Input;
			Input = Output;
			Output = Temp;

			//Multiply
			hipblasSgemv(Handle, HIPBLAS_OP_N,
				HiddenNeurons, HiddenNeurons, &Alpha,
				Matrix, HiddenNeurons, Input, 1, &Beta, Output, 1);
			hipDeviceSynchronize();

			Matrix += HiddenNeurons * HiddenNeurons;

			//Add biases
			hipblasSaxpy(Handle, HiddenNeurons,
				&Alpha, Matrix, 1, Output, 1);
			hipDeviceSynchronize();

			//Activation function
			ReLU <<<(HiddenNeurons - 1) / 32 + 1, 32>>> (Output, HiddenNeurons);
			hipDeviceSynchronize();

			Matrix += HiddenNeurons;
		}

		//Hidden output multiplication + biases
		//Multiply
		hipblasSgemv(Handle, HIPBLAS_OP_N,
			OutputNeurons, HiddenNeurons, &Alpha,
			Matrix, OutputNeurons, Input, 1, &Beta, Output, 1);
		hipDeviceSynchronize();

		Matrix += HiddenNeurons * OutputNeurons;

		//Add biases
		hipblasSaxpy(Handle, OutputNeurons,
			&Alpha, Matrix, 1, Output, 1);
		hipDeviceSynchronize();

		//Activation function
		Sigmoid <<<(OutputNeurons - 1) / 32 + 1, 32>>> (Output, OutputNeurons);
		hipDeviceSynchronize();

		hipMemcpy(&TempFitness, Output, sizeof(float), hipMemcpyDeviceToHost);
		Fitness += (OutputFeatures[j] - TempFitness) * (OutputFeatures[j] - TempFitness);
	}

	Fitness /= NumSamples;

	cout << "TEST FITNESS: " << Fitness << endl;
}
